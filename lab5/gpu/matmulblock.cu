
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <chrono>
#define N (1 << 10)
#define BLOCK_SIZE 16
#include <stdlib.h>


__global__ void gemm_block(float *A, float *B, float *C);
void gemm_verify(float *A, float *B, float *C);

int main()
{
	// malloc A, B, C
	float *A = (float*)malloc(N * N * sizeof(float));
	float *B = (float*)malloc(N * N * sizeof(float));
	float *C = (float*)malloc(N * N * sizeof(float));

	// random initialize A, B
	for (int i = 0; i < N * N; i++) {
		A[i] = (float)rand() / RAND_MAX;
		B[i] = (float)rand() / RAND_MAX;
		C[i] = 0;
	}

	// cumalloc A, B, C
	float *cuda_A, *cuda_B, *cuda_C;
	hipMalloc(&cuda_A, N * N * sizeof(float));
	hipMalloc(&cuda_B, N * N * sizeof(float));
	hipMalloc(&cuda_C, N * N * sizeof(float));

	hipMemcpy(cuda_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cuda_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cuda_C, C, N * N * sizeof(float), hipMemcpyHostToDevice);

	// define gridsize and blocksize
	dim3 blocksize(BLOCK_SIZE, BLOCK_SIZE);
	dim3 gridsize((N + blocksize.x - 1) / blocksize.x, (N + blocksize.y - 1) / blocksize.y);

	// compute
	auto start = std::chrono::high_resolution_clock::now();
	gemm_block<<<gridsize, blocksize>>>(cuda_A, cuda_B, cuda_C);
	auto end = std::chrono::high_resolution_clock::now();
	hipDeviceSynchronize();

	std::chrono::duration<double> diff = end - start;
	printf("Time2: %f s\n", diff.count());

	hipMemcpy(C, cuda_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

	// gemm_verify(A, B, C);
	gemm_verify(A, B, C);

	// free mem
	hipFree(cuda_A);
	hipFree(cuda_B);
	hipFree(cuda_C);
	free(A);
	free(B);
	free(C);

	return 0;
}

__global__ void gemm_block(float* A, float * B, float* C) {
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
    int srow = threadIdx.y;
    int scol = threadIdx.x;

    for (int b = 0; b < N/BLOCK_SIZE; b++) {
        As[srow][scol] = A[row * N + b * BLOCK_SIZE + scol];
        Bs[srow][scol] = B[(b * BLOCK_SIZE + srow) * N + col];
        __syncthreads();
        for (int i = 0; i < BLOCK_SIZE; i++) {
            C[row * N + col] += As[srow][i] * Bs[i][scol];
        }
        __syncthreads();
    }
}

void gemm_verify(float *A, float *B, float *C)
{
    float *baseline = (float *)malloc(N * N * sizeof(float));
    for (int i = 0; i < N * N; i++)
        baseline[i] = 0;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            for (int k = 0; k < N; k++)
            {
                baseline[i * N + j] += A[i * N + k] * B[k * N + j];
            }
        }
    }

    for (int i = 0; i < N * N; i++)
    {
        if (C[i] - baseline[i]>1e-3|| C[i] - baseline[i]<-1e-3)
        {
            printf("fail: C[%d] = %f, baseline[%d] = %f\n", i, C[i], i, baseline[i]);
            break;
        }
    }
    free(baseline);
}
